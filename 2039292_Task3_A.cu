#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//__global__ --> GPU function which can be launched by many blocks and threads
//__device__ --> GPU function or variables
//__host__ --> CPU function or variables

__device__ char* CudaCrypt(char* rawPassword){

	char * newPassword = (char *) malloc(sizeof(char) * 11);

	newPassword[0] = rawPassword[0] + 2;  
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;  
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword; //Returns encrypted password
}

__device__ int compareTwoString(char* stringOne, char* stringTwo){
	
    while(*stringOne)
    {
        //Comparing the two strings
        if (*stringOne != *stringTwo)
            break;
 
        //Changing Pointer location
        stringOne++;
        stringTwo++;
    }
 
    // Returing the 0 if the two strings matches 
    return *(const unsigned char*)stringOne - *(const unsigned char*)stringTwo;
}

__global__ void crack(char * alphabet, char * numbers, char * rawPassword){

char genRawPass[4];
//Adding test passwords to genRawPass
genRawPass[0] = alphabet[blockIdx.x];
genRawPass[1] = alphabet[blockIdx.y];

genRawPass[2] = numbers[threadIdx.x];
genRawPass[3] = numbers[threadIdx.y];

//Raw Password being encrypted
char *encPassword = CudaCrypt(rawPassword);
	
	//Comparing encrypted genRawPass with encPassword
	if(compareTwoString(CudaCrypt(genRawPass),encPassword) == 0){
		printf("Your password is cracked : %s = %s\n", genRawPass, rawPassword);
	}
}

int time_difference(struct timespec *start, struct timespec *finish, long long int *difference){
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

int main(int argc, char ** argv){

char cpuAlphabet[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
char cpuNumbers[10] = {'0','1','2','3','4','5','6','7','8','9'};

char * gpuAlphabet;
hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26); 
hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

char * gpuNumbers;
hipMalloc( (void**) &gpuNumbers, sizeof(char) * 10); 
hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 10, hipMemcpyHostToDevice);

char * password;
hipMalloc( (void**) &password, sizeof(char) * 26); 
hipMemcpy(password, argv[1], sizeof(char) * 26, hipMemcpyHostToDevice);

	struct timespec start, finish;
	long int time_elapsed;
	
//Start monitoring the duration 
	clock_gettime(CLOCK_MONOTONIC, &start);
	
	crack<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuAlphabet, gpuNumbers, password);
	hipDeviceSynchronize();

//End the duration of the program
	clock_gettime(CLOCK_MONOTONIC, &finish);
	
//Calculate the duration
	time_difference(&start, &finish, &time_elapsed);
	
//Print the duration taken
	printf(" Time taken to crack : %lld",time_elapsed);
return 0;
}


	











